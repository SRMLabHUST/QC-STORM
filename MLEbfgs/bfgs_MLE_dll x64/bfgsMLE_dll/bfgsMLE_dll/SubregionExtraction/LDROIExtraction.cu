/*
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU LESSER GENERAL PUBLIC LICENSE as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU LESSER GENERAL PUBLIC LICENSE for more details.

You should have received a copy of the GNU LESSER GENERAL PUBLIC LICENSE
along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

#include "LDROIExtraction.h"


#include "bfgsMLE_core.h"

#include <math.h>


#define Max(a,b)    (((a) > (b)) ? (a) : (b))
#define Min(a,b)    (((a) < (b)) ? (a) : (b))




void LDROIExtractData_TypeDef::ExtractMolecules(unsigned short *pImgData, int ImageSource, LocalizationPara & LocPara, int StartFrame_CurBatch, int BatchedImageNum, hipStream_t cstream)
{
	const int BatchedImageHigh = BatchedImageNum * LocPara.ImageHigh;

	const int BatchImagePixelNum = BatchedImageNum * LocPara.ImageWidth * LocPara.ImageHigh;


	// first batch, localization may be perfomed after multi extraction batches
	if ((FirstFrame < 0) && (EndFrame < 0))
	{
		FirstFrame = StartFrame_CurBatch;
		EndFrame = FirstFrame + BatchedImageNum - 1;
	}
	else
	{
		EndFrame = EndFrame + BatchedImageNum;
	}

	// get raw image
	if (ImageSource == ImageSource_CPU_Pinned)
	{
		// image from CPU
		hipMemcpyAsync(d_RawImg, pImgData, BatchImagePixelNum * sizeof(unsigned short), hipMemcpyHostToDevice, cstream); // h_RawImg
	}
	else if (ImageSource == ImageSource_CPU_Normal)
	{
		// image from CPU
		hipMemcpy(d_RawImg, pImgData, BatchImagePixelNum * sizeof(unsigned short), hipMemcpyHostToDevice); // h_RawImg
	}
	else if (ImageSource == ImageSource_GPU)
	{
		// always cause problem
		//		hipMemcpy(d_RawImg, pImgData, BatchImagePixelNum * sizeof(unsigned short), hipMemcpyDeviceToDevice); // h_RawImg
		//		hipMemcpyAsync(d_RawImg, pImgData, BatchImagePixelNum * sizeof(unsigned short), hipMemcpyDeviceToDevice, cstream);

		printf("data error\n");
		return;
	}
	else
	{
		printf("data error\n");
		// image data error
		return;
	}

	// ImageFiltering
	ImageFiltering(LocPara.LocType, LocPara.MultiEmitterFitEn, LocPara.ImageWidth, LocPara.ImageHigh, BatchedImageNum, cstream);

	ROIExtraction(LocPara.ROISize, LocPara.LocType, LocPara.MultiEmitterFitEn, LocPara.ImageWidth, LocPara.ImageHigh, BatchedImageNum, StartFrame_CurBatch, cstream);

}


void LDROIExtractData_TypeDef::ROIExtraction(int ROISize, int LocType, int MultiEmitterFitEn, int ImageWidth, int ImageHigh, int BatchedImageNum, int StartFrame, hipStream_t cstream)
{
	const int BatchedImageHigh = BatchedImageNum*ImageHigh;
	const int ROIWholeSize = ROISize*(ROISize + 1);


	hipMemsetAsync(d_ValidROINum, 0, sizeof(int), cstream);

	hipMemsetAsync(d_MoleculePosImage, 0, ImageWidth * BatchedImageHigh * sizeof(short), cstream);

	// find possible ROI position

	ROIFindingLD(d_RawImg_Smoothed, d_MoleculePosImage, d_ValidROINum, d_ROIMarkInfArray, ImageVariance, LocType, MultiEmitterFitEn, ROISize, ImageWidth, BatchedImageHigh, ImageHigh, cstream);


	hipMemcpyAsync(h_ValidROINum, d_ValidROINum, sizeof(int), hipMemcpyDeviceToHost, cstream);
	hipStreamSynchronize(cstream);

	int ROINumber = *h_ValidROINum;


	// pre-clasify molecule by single or multi by ROI molecule distance

	DetectedROIClasify(d_MoleculePosImage, d_ROIMarkInfArray, ROINumber, MultiEmitterFitEn, ROISize, ImageWidth, cstream);

	hipStreamSynchronize(cstream);
	
	int ROINum_CurBatch = *h_ValidROINum;


	// ROI extraction

	SubregionExtractionLD(d_RawImg, d_ImageROI, d_ROIMarkInfArray, ROISize, ROINum_CurBatch, ImageWidth, BatchedImageHigh, ImageHigh, StartFrame, cstream);


	int ROIAddrOffset = TotalROINumber*ROIWholeSize;

	hipMemcpyAsync(&h_ImageROI[ROIAddrOffset], d_ImageROI, ROINum_CurBatch * ROIWholeSize * sizeof(short), hipMemcpyDeviceToHost, cstream);
	hipStreamSynchronize(cstream); // wait task of this stream finish


#if(WLE_ENABLE == 1)

	// estimate WLE parameter
	int WLEParaAddrOffset = TotalROINumber*WLE_ParaNumber;

	WLEParameterEstimator->WLEParameterEstimate(d_ImageROI, LocType, MultiEmitterFitEn, ROISize, ROINum_CurBatch, cstream);// &h_ImageROI[ROIAddrOffset]

	hipMemcpyAsync(&WLEParameterEstimator->h_WLEPara[WLEParaAddrOffset], WLEParameterEstimator->d_WLEPara, ROINum_CurBatch * WLE_ParaNumber * sizeof(float), hipMemcpyDeviceToHost, cstream);
	hipStreamSynchronize(cstream); // wait task of this stream finish

#endif // WLE_ENABLE

	TotalROINumber += ROINum_CurBatch;

}

void LDROIExtractData_TypeDef::ImageVarianceCalc(unsigned short *d_iRawImg, int ImageWidth, int ImageHigh, hipStream_t cstream)
{
	int ImageCenterX = ImageWidth / 2;
	int ImageCenterY = ImageHigh / 2;

	int StdImageWidth = ImageWidth / 2 - 5;
	int StdImageHigh = ImageHigh / 2 - 5;


	StdImageWidth = Min(StdImageWidth, 256);
	StdImageHigh = Min(StdImageHigh, 256);


	StdImageWidth = Max(StdImageWidth, 1);
	StdImageHigh = Max(StdImageHigh, 1);


	int StdImageSize = Min(StdImageWidth, StdImageHigh);

	hipMemsetAsync(d_MeanDataX, 0, sizeof(float), cstream);
	hipMemsetAsync(d_MeanDataX2, 0, sizeof(float), cstream);

	ImageVarianceCalcOnSelectRegion(d_iRawImg, d_MeanDataX, d_MeanDataX2, ImageCenterX, ImageCenterY, StdImageSize, ImageWidth, ImageHigh, cstream);


	hipMemcpyAsync(h_MeanDataX, d_MeanDataX, sizeof(float), hipMemcpyDeviceToHost, cstream);
	hipMemcpyAsync(h_MeanDataX2, d_MeanDataX2, sizeof(float), hipMemcpyDeviceToHost, cstream);

	hipStreamSynchronize(cstream);


	ImageVariance = *h_MeanDataX2 - (*h_MeanDataX)*(*h_MeanDataX);

	if (ImageVariance < 100.0f) ImageVariance = 100.0f;


	//	printf("ImageVariance:%f\n", ImageVariance);
}



int LDROIExtractData_TypeDef::GetAccumulatedROINum()
{
	return TotalROINumber;
}


void LDROIExtractData_TypeDef::ResetROINum()
{
	TotalROINumber = 0;

	FirstFrame = -1;
	EndFrame = -1;

}

float * LDROIExtractData_TypeDef::Get_h_WLEPara()
{
	return WLEParameterEstimator->h_WLEPara;
}

float * LDROIExtractData_TypeDef::Get_d_WLEPara()
{
	return WLEParameterEstimator->d_WLEPara;

}

int LDROIExtractData_TypeDef::GetMaxBatchedNumForCurrentImageSize(int ImageWidth, int ImageHigh)
{
	int MaxBatchImgNum = (2048 * 2 / (ImageWidth + ImageHigh));

	MaxBatchImgNum = MaxBatchImgNum / 2 * 2;

	if (MaxBatchImgNum < 1)MaxBatchImgNum = 1;

	return MaxBatchImgNum;
}


void LDROIExtractData_TypeDef::Init(LocalizationPara & LocPara)
{
	hipError_t err;


	int MaxBatchImgNum = GetMaxBatchedNumForCurrentImageSize(LocPara.ImageWidth, LocPara.ImageHigh);

	WLEParameterEstimator = new WLEParameterEstimation_TypeDef();
	WLEParameterEstimator->Init(LocPara);


	// raw image
	err = hipHostMalloc((void **)&h_RawImg, MaxBatchedImageSize * sizeof(short));
	HandleErr(err, "hipMalloc h_RawImg");

	err = hipMalloc((void **)&d_RawImg, MaxBatchedImageSize * sizeof(short));
	HandleErr(err, "hipMalloc d_RawImg");

	err = hipMalloc((void **)&d_RawImg_Smoothed, MaxBatchedImageSize * sizeof(short));
	HandleErr(err, "hipMalloc d_RawImg_Smoothed");


	err = hipMalloc((void **)&d_BackgroundImage, MaxBatchedImageSize * sizeof(short));
	HandleErr(err, "hipMalloc d_BackgroundImage");

	err = hipMalloc((void **)&d_LineFilterImage_t, MaxBatchedImageSize * sizeof(short));
	err = hipMalloc((void **)&d_LineFilterImage_t1, MaxBatchedImageSize * sizeof(short));


	err = hipMalloc((void **)&d_MoleculePosImage, MaxBatchedImageSize * sizeof(short));

	// image std use

	err = hipHostMalloc((void **)&h_MeanDataX, sizeof(float));
	err = hipHostMalloc((void **)&h_MeanDataX2, sizeof(float));

	err = hipMalloc((void **)&d_MeanDataX, sizeof(float));
	err = hipMalloc((void **)&d_MeanDataX2, sizeof(float));


	// image filter

	//	err = hipHostMalloc((void **)&h_LineFilterH_Bkg, LineFilterSize * sizeof(float));
	//	err = hipMalloc((void **)&d_LineFilterH_Bkg, LineFilterSize * sizeof(float));

	err = hipHostMalloc((void **)&h_LineFilterH_Signal, LineFilterSize * sizeof(float));
//	err = hipMalloc((void **)&d_LineFilterH_Signal, LineFilterSize * sizeof(float));


	// extracted molecular ROI data
	const int ROIWholeSize = LocPara.ROISize*(LocPara.ROISize + 1);

	err = hipHostMalloc((void **)&h_ImageROI, MaxPointNum * ROIWholeSize * sizeof(unsigned short));
	err = hipMalloc((void **)&d_ImageROI, MaxPointNum * ROIWholeSize * sizeof(unsigned short));

	err = hipHostMalloc((void **)&h_ValidROINum, sizeof(int));
	err = hipMalloc((void **)&d_ValidROINum, sizeof(int));


	err = hipMalloc((void **)&d_ROIMarkInfArray, MaxPointNum * ROIMarkInfNum * sizeof(int));


	//
	FilterInit(LocPara.ROISize, LocPara.LocType, LocPara.MultiEmitterFitEn);

	ResetROINum();

}


void LDROIExtractData_TypeDef::Deinit()
{
	hipError_t err;

	WLEParameterEstimator->Deinit();

	delete WLEParameterEstimator;


	// raw image filtering
	err = hipHostFree(h_RawImg);
	HandleErr(err, "hipHostFree h_RawImg");

	err = hipFree(d_RawImg);
	HandleErr(err, "hipFree d_RawImg");

	err = hipFree(d_RawImg_Smoothed);


	err = hipFree(d_BackgroundImage);
	HandleErr(err, "hipFree d_BackgroundImage");

	err = hipFree(d_LineFilterImage_t);
	err = hipFree(d_LineFilterImage_t1);

	err = hipFree(d_MoleculePosImage); //�޸� zzw 20200915
	// image std use

	err = hipHostFree(h_MeanDataX);
	err = hipHostFree(h_MeanDataX2);

	err = hipFree(d_MeanDataX);
	err = hipFree(d_MeanDataX2);


	// image filter
	//	err = hipHostFree(h_LineFilterH_Bkg);
	//	err = hipFree(d_LineFilterH_Bkg);

	err = hipHostFree(h_LineFilterH_Signal);
//	err = hipFree(d_LineFilterH_Signal);


	// extracted molecular ROI data
	err = hipHostFree(h_ImageROI);
	err = hipFree(d_ImageROI);

	err = hipHostFree(h_ValidROINum);
	err = hipFree(d_ValidROINum);

	err = hipFree(d_ROIMarkInfArray);

}


